#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

__global__ void opt_2dhistoKernel(uint32_t *input[], size_t height, size_t width, uint8_t bins[HISTO_HEIGHT*HISTO_WIDTH]);

void opt_2dhisto(uint32_t *input[], size_t height, size_t width, uint8_t bins[HISTO_HEIGHT*HISTO_WIDTH])
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */

    hipMemset	(bins, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(bins[0]));

    opt_2dhistoKernel<<<1, 1024>>>(input, height, width, bins);
}

/* Include below the implementation of any other functions you need */

__global__ void opt_2dhistoKernel(uint32_t *input[], size_t height, size_t width, uint8_t bins[HISTO_HEIGHT*HISTO_WIDTH]){

    for (size_t j = 0; j < height; ++j)
    {
        for (size_t i = 0; i < width; ++i)
        {
            const uint32_t value = input[j][i];

            uint8_t *p = (uint8_t*)bins;

            // Increment the appropriate bin, but do not roll-over the max value
            if (p[value] < UINT8_MAX)
                ++p[value];
        }
    }
}

void* AllocateDevice(size_t size){
	void* ret;
	hipMalloc(ret, size);
	return ret;
}

void CopyToDevice(void* D_device, void* D_host, size_t size){
	hipMemcpy(D_device, D_host, size, 
					hipMemcpyHostToDevice);
}

void CopyFromDevice(void* D_host, void* D_device, size_t size){
	hipMemcpy(D_host, D_device, size, 
					hipMemcpyDeviceToHost);
}

void FreeDevice(void* D_device){
	hipFree(D_device);
}