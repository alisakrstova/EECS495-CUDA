#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

#define T 12

__global__ void opt_2dhistoKernel(uint32_t *input, size_t height, size_t width, uint32_t* bins);
__global__ void opt_32to8Kernel(uint32_t *input, uint8_t* output, size_t length);

void opt_2dhisto(uint32_t* input, size_t height, size_t width, uint8_t* bins)
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */
	uint32_t* g_bins;
	hipMalloc(&g_bins, HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint32_t));

    hipMemset(bins, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(bins[0]));
    hipMemset(g_bins, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(g_bins[0]));
    
    
    // Kernel to calculate the bins
    // We use 1024 * T threads so that more streaming multiprocessors can be used
    opt_2dhistoKernel<<<2 * T, 512>>>(input, height, width, g_bins);
  
    // Convert 332 bit to 8 bit
    opt_32to8Kernel<<<HISTO_HEIGHT * HISTO_WIDTH / 512, 512>>>(g_bins, bins, 1024);

    hipDeviceSynchronize();
    hipFree(g_bins);
}

/* Include below the implementation of any other functions you need */

__global__ void opt_2dhistoKernel(uint32_t *input, size_t height, size_t width, uint32_t* bins){
	
	// Shared memory to hold the sub-histogram
	__shared__ int sub_hist[1024];
	
	// No bank conflict as threads access the shared memory sequentially
	sub_hist[threadIdx.x] = 0;
	sub_hist[threadIdx.x + 512] = 0;
	
	// Global thread id
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
		
	// Step 1: Divide each row into different groups so that each thread can concurrently work on a portion of the row
	// 	   Acheived by dividing the row into 'T' groups each having (width/T) elements.
	// Step 2: Calculate the row id and column id according to this new configuration
	// 	   Row id = (idx / T) * height of the input data
	// 	   Column id = (idx % T) * (width / T) , as each thread accesses data strided in the columns by (width / T)
	// Step 3: Each thread performs work only in its assigned group
	// 	   The loop runs from 0 to (width / T)
	
	for (int i = 0; i < (width / T); ++i)
	{
		atomicAdd(sub_hist + input[((idx / T) * height) + ((idx % T) * (width / T)) +  i], 1);
	}
	__syncthreads();
	
	// Update the global memory - storing is again sequential so no bank conflict
	atomicAdd(bins + threadIdx.x, sub_hist[threadIdx.x]);
	atomicAdd(bins + threadIdx.x + 512, sub_hist[threadIdx.x + 512]);
}

__global__ void opt_32to8Kernel(uint32_t *input, uint8_t* output, size_t length){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	output[idx] = (uint8_t)((input[idx] < UINT8_MAX) * input[idx]) + (input[idx] >= UINT8_MAX) * UINT8_MAX;

	__syncthreads();
}

void* AllocateDevice(size_t size){
	void* ret;
	hipMalloc(&ret, size);
	return ret;
}

void CopyToDevice(void* D_device, void* D_host, size_t size){
	hipMemcpy(D_device, D_host, size, 
					hipMemcpyHostToDevice);
}

void CopyFromDevice(void* D_host, void* D_device, size_t size){
	hipMemcpy(D_host, D_device, size, 
					hipMemcpyDeviceToHost);
}

void FreeDevice(void* D_device){
	hipFree(D_device);
}