#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

__global__ void opt_2dhistoKernel(uint32_t *input, size_t height, size_t width, uint8_t* bins);

void opt_2dhisto(uint32_t* input, size_t height, size_t width, uint8_t* bins)
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */

    hipMemset	(bins, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(bins[0]));

    opt_2dhistoKernel<<<1, 512>>>(input, height, width, bins);

    hipDeviceSynchronize();
}

/* Include below the implementation of any other functions you need */

__global__ void opt_2dhistoKernel(uint32_t *input, size_t height, size_t width, uint8_t* bins){

    int idx = threadIdx.x;
    __shared__ uint s_bins[HISTO_HEIGHT*HISTO_WIDTH];


	if (s_bins[input[idx]] < UINT8_MAX)
		atomicAdd(s_bins + input[idx], 1);
		//++bins[input[j * height + idx]];
	if (s_bins[input[idx + width / 2]] < UINT8_MAX)
		atomicAdd(s_bins + input[idx + width / 2], 1);
		//++bins[input[j * height + idx + width / 2]];

    __syncthreads();
    bins[idx] = (uint8_t)s_bins[idx];
    __syncthreads();
    bins[idx + width / 2] = (uint8_t)s_bins[idx + width / 2];
    __syncthreads();
}

void* AllocateDevice(size_t size){
	void* ret;
	hipMalloc(&ret, size);
	return ret;
}

void CopyToDevice(void* D_device, void* D_host, size_t size){
	hipMemcpy(D_device, D_host, size, 
					hipMemcpyHostToDevice);
}

void CopyFromDevice(void* D_host, void* D_device, size_t size){
	hipMemcpy(D_host, D_device, size, 
					hipMemcpyDeviceToHost);
}

void FreeDevice(void* D_device){
	hipFree(D_device);
}